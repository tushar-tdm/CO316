#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
    
#define TILE_WIDTH 32

#define printError(func)                                                \
{                                                                       \
  hipError_t E  = func;                                                \
  if(E != hipSuccess)                                                  \
  {                                                                     \
    printf( "\nError at line: %d ", __LINE__);                          \
    printf( "\nError:  %s ", hipGetErrorString(E));                    \
  }                          `                                           \
}                                                                       \ 

__global__ void TiledMatrixMult(int m, int n, int k, int *A, int *B, int *C)
{
    int CValue = 0;

    int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;

    __shared__ int As[TILE_WIDTH][TILE_WIDTH];
    __shared__ int Bs[TILE_WIDTH][TILE_WIDTH];

    for (int i = 0; i < (TILE_WIDTH + n - 1)/TILE_WIDTH; i++) {

         if (i*TILE_WIDTH + threadIdx.x < n && Row < m)
             As[threadIdx.y][threadIdx.x] = A[Row*n + i*TILE_WIDTH + threadIdx.x];
         else
             As[threadIdx.y][threadIdx.x] = 0.0;

         if (i*TILE_WIDTH + threadIdx.y < n && Col < k)
             Bs[threadIdx.y][threadIdx.x] = B[(i*TILE_WIDTH + threadIdx.y)*k + Col];
         else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int j = 0; j < TILE_WIDTH; ++j)
             CValue += As[threadIdx.y][j] * Bs[j][threadIdx.x];

         __syncthreads();
    }

    if (Row < m && Col < k)
        C[((blockIdx.y * blockDim.y + threadIdx.y)*k) +
           (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
}

int checkProd(int m, int n, int k, int *A, int *B, int*C)
{
    for(int row= 0;row<m;row++)
    {
        for(int col=0;col<k;col++)
        {
            int sum=0;
            for(int i=0;i<n;i++)
            {
                sum = sum + A[row*n + i] * B[col + i*k];
            }


            if(C[row*k + col] != sum)
                return 0;
        }
    }
    return 1;
}


int main()
{
    int *A;
    int *B;
    int *C;

    int *deviceA;
    int *deviceB;
    int *deviceC;

    // Matrix A of size (m,n) and Matrix B of size (n,k)
    int m = 1024;
    int n = 512;
    int k = 1024;

    A = (int *)malloc(m * n * sizeof(int));
    B = (int *)malloc(n * k * sizeof(int));
    C = (int *)malloc(m * k * sizeof(int));

    for(int i=0;i<m*n;i++)
    {
        A[i] = rand()%10;
        //printf("%d ",A[i]);
    }
    printf("\n");

    for(int i=0;i<n*k;i++)
    {
        B[i] = rand()%10;
        //printf("%d ",B[i]);
    }
    //printf("\n");

    hipMalloc((void **)&deviceA,  m * n * sizeof(int));
    hipMalloc((void **)&deviceB,  n * k * sizeof(int));
    hipMalloc((void **)&deviceC,  m * k * sizeof(int));

    hipMemcpy(deviceA, A, m * n *  sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B, n * k *  sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid((k-1)/TILE_WIDTH+1, (m-1)/TILE_WIDTH+1,1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    //dim3 dimGrid(32,32,1);
    //dim3 dimBlock(32,32,1);

    TiledMatrixMult<<<dimGrid, dimBlock>>>(m,n,k,deviceA,deviceB,deviceC);

    hipMemcpy(C, deviceC, m * k * sizeof(float), hipMemcpyDeviceToHost);

    if(checkProd(m, n, k, A, B, C))
      printf("\nResult of matrix multiplication is correct\n");

    else
       printf("\nResult of matrix multiplication is wrong\n");

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    free(A);
    free(B);
    free(C);
}
